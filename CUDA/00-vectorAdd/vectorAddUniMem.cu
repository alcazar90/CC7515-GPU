#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ 
void vectorAddition(float* a, float* b, float* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = b[index] + a[index];
    }
}


int main() {
    int size = 1 << 20;
    float *a, *b, *c;

    // Allocate Unified Memory -- accessible from CPU or GPU
    // Look more details in https://devblogs.nvidia.com/unified-memory-cuda-beginners/
    hipMallocManaged(&a, size * sizeof(float));
    hipMallocManaged(&b, size * sizeof(float));
    hipMallocManaged(&c, size * sizeof(float));

    // Initialize input vectors
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Set up grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel on the GPU
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, size);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Print the result vector
    //for (int i = 0; i < size; i++) {
        ///std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    //}

    // No need to call cudaFree for a, b, c

    return 0;
}
