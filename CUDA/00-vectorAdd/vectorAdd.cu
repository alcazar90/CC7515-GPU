#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ 
void vectorAddition(float* a, float* b, float* c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < size) {
        c[index] = b[index] + a[index];
    }
}

int main() {
    int size = 20;
    float* a = new float[size];
    float* b = new float[size];
    float* c = new float[size];

    // Initialize input vectors
    for (int i =0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    // Allocate memory
    float* d_a, * d_b, * d_c;
    hipMalloc((void **) &d_a, size * sizeof(float));
    hipMalloc((void **) &d_b, size * sizeof(float));
    hipMalloc((void **) &d_c, size * sizeof(float));

    // Copy input vectors from host memory to GPU buffers
    hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

    // Set up grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel on the GPU
    vectorAddition<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Copy the result vector from device to host
    hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result vector
    for (int i = 0; i < size; i++) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }

    // Free memory on the device
    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c);

    // Free memoruy on the host
    delete[] a; 
    delete[] b; 
    delete[] c;

    return 0;
}
