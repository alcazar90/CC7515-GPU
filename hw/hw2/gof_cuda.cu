#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <chrono>
#include <iostream>
#include <fstream>
#include <vector>
#include <random>
#include <cstddef>
#include <cassert>
#include "include/utils.hpp"

// Function prototypes
typedef unsigned char ubyte;
std::string toString2D(const ubyte* grid, int nRows, int nCols);
void initializeBoardFromFile(ubyte* grid, int nRows, int nCols, const std::string& filename);
void initializeBoardRandom(ubyte* grid, int nRows, int nCols);


__global__ void gameOfLifeKernel(const ubyte* grid, int nRows, int nCols, 
                                 ubyte* result) {
    // Base on the following implementation.
    // Source: http://www.marekfiser.com/Projects/Conways-Game-of-Life-on-GPU-using-CUDA/2-Basic-implementation#code-listing-2
    uint gridSize = nRows * nCols;

    for (uint cellId = (blockIdx.x * blockDim.x) + threadIdx.x;
         cellId < gridSize;
         cellId += blockDim.x * gridDim.x) {
            uint x = cellId % nCols;
            uint yAbs = cellId - x;
            uint xLeft = (x + nCols - 1) % nCols;
            uint xRight = (x + 1) % nCols;
            uint yAbsUp = (yAbs + gridSize - nCols) % gridSize;
            uint yAbsDown = (yAbs + nCols) % gridSize;

            // Shrink count neighbors using if statements
            uint aliveCells = grid[xLeft + yAbsUp] + grid[x + yAbsUp]
                + grid[xRight + yAbsUp] + grid[xLeft + yAbs] + grid[xRight + yAbs]
                + grid[xLeft + yAbsDown] + grid[x + yAbsDown] + grid[xRight + yAbsDown];
            
            result[x + yAbs] =
                aliveCells == 3 || (aliveCells == 2 && grid[x + yAbs]) ? 1 : 0;
         }
}


/* Main Program */
int main() {

    // Inicializador - Configuracion de parámetros del juego
    initGameConfig();

    // Allocate memory for the game grid
    uint gridSize = NROWS * NCOLS;
    ubyte* grid;
    ubyte* resultGrid;
    hipMallocManaged(&grid, gridSize * sizeof(ubyte));
    hipMallocManaged(&resultGrid, gridSize * sizeof(ubyte));

    if (filename == "RANDOM") {
        initializeBoardRandom(grid, NROWS, NCOLS);
    } else if (filename.find(".txt") != std::string::npos) {
        initializeBoardFromFile(grid, NROWS, NCOLS, filename);
    } else {
        std::cout << "Invalid input. Exiting..." << std::endl;
        return 0;
    }

    std::cout << "Welcome to the game of life. We will play " << NITER << " generations...and the universe will be a grid of size " << NROWS << "x" << NCOLS << std::endl;
    std::cout << "Here is your initial board configuration:\n";
    std::cout << toString2D(grid, NROWS, NCOLS) << std::endl;

    // CUDA config
    ushort threadsCount = NCOLS * NROWS;
    assert((NCOLS * NROWS) % threadsCount == 0);
    size_t reqBlocksCount = (NCOLS * NROWS) / threadsCount;
    ushort blocksCount = (ushort)std::min((size_t)32768, reqBlocksCount);

    // Calculate the time it takes for the simulation
    double itime = 0.0;
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < NITER; ++i) {
        // Print the grid
        if (PRETTYPRINT) {
            std::cout << "Generation " << (i + 1) << ":" << std::endl;
            std::cout << toString2D(grid, NROWS, NCOLS) << std::endl;
        }

        // Launch the kernel
        gameOfLifeKernel<<<blocksCount, threadsCount>>>(grid, NROWS, NCOLS, resultGrid);

        // Wait for GPU to finish before processing the following generation
        hipDeviceSynchronize();

        // Swap values of two objects efficiently
        std::swap(grid, resultGrid);

    }

    auto end = std::chrono::steady_clock::now();
    itime = (float)std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    // Print the total time in seconds
    std::cout << "---------------------------------------" << std::endl;
    std::cout << "Total time: " << itime / 1000000000.0 << " seconds" << std::endl;

    return 0;
}

std::string toString2D(const ubyte* grid, int nRows, int nCols) {
    std::string result;
    for (int i = 0; i < nRows; ++i) {
        for (int j = 0; j < nCols; ++j) {
            int index = i * nCols + j;
            result += (grid[index] == 1) ? "1 " : ". ";
        }
        result += "\n";
    }
    return result;
}

void initializeBoardRandom(ubyte* grid, int nRows, int nCols) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 1);

    for (int i = 0; i < nRows; ++i) {
        for (int j = 0; j < nCols; ++j) {
            int index = i * nCols + j;
            grid[index] = dis(gen);
        }
    }
}

void initializeBoardFromFile(ubyte* grid, int nRows, int nCols, const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        throw std::runtime_error("Error opening file: " + filename);
    }

    uint value;
    for (int i = 0; i < nRows; ++i) {
        for (int j = 0; j < nCols; ++j) {
            if (file >> value) {
                int index = i * nCols + j;
                grid[index] = value;
            } else {
                throw std::runtime_error("Error reading file: " + filename);
            }
        }
    }

    file.close();
}
